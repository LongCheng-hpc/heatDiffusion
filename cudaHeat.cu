#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include "iostream"
#include "outMatrix.h"
#include "calModuls.h"
#include "initMatrix.h"
#define BLOCK_SIZE 16
/* TODO：
 * 1. 读取运行脚本指令，来设置所需要的参数
 * 2. 读取目标文件，来初始化数组
 *      - 第一行为rows和cols
 *      - 第二行为初始点的坐标以及初始值（float）
 * 3. 计算每一步的热量传递
 * 4. 计算平均温度
 * 5. 进行输出
 * add an outer layer of 0
*/

__global__ void gpu_diffuse(const float * nowMatrix, float * nextMatrix, int rows, int cols, float constant)
{
    // every block share same memory
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int wholeCol = cols + 2;
    float tmpVal = nowMatrix[row * wholeCol + col];

    if(0 < row && row <= rows && col > 0 && col <= cols)
    {
        float surrounding =  (nowMatrix[row * wholeCol + col - 1] +
                        nowMatrix[row * wholeCol + col + 1] +
                        nowMatrix[(row - 1) * wholeCol  + col] +
                        nowMatrix[(row + 1) * wholeCol  + col]) / 4 - tmpVal;

        float nextValue =  tmpVal + constant * surrounding;

        nextMatrix[row * wholeCol + col] = nextValue;
    } 
    __syncthreads();
}


int main(int argc, char *argv[]) {
// 修改为argv[0]以及转换数据类型
    int iterations = 1000;
    float constant = 0.02;

    const int rows = 10000;
    const int cols = 10000;

    int initX = 5000;
    int initY = 5000;

    float initVal = 10000000000;

    float *nowMatrix;
    float *nextMatrix;

    // init host
    nowMatrix = (float*) malloc(sizeof(float) * (rows + 2) * (cols + 2));
    nextMatrix = (float*) malloc(sizeof(float) * (rows + 2) * (cols + 2));

    nowMatrix[(initX + 1) * (cols + 2) + initY + 1] = initVal;

    // init device
    float * cudaNow, *cudaNext;
    hipMalloc((void**)&cudaNow, sizeof(float) * (rows + 2) * (cols + 2));
    hipMalloc((void**)&cudaNext, sizeof(float) * (rows + 2) * (cols + 2));

    hipMemcpy(cudaNow, nowMatrix, sizeof(float) * (rows + 2) * (cols + 2), hipMemcpyHostToDevice);

    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // gpu_diffuse <<<dimGrid, dimBlock>>> (cudaNow, cudaNext, rows, cols, constant);

    hipMemcpy(nextMatrix, cudaNext, sizeof(float) * (rows + 2) * (cols + 2), hipMemcpyDeviceToHost);
    
    for(int it = 0; it < iterations; it++){
        gpu_diffuse <<<dimGrid, dimBlock>>> (cudaNow, cudaNext, rows, cols, constant);
        hipDeviceSynchronize();
        cudaNow = cudaNext;
    }
    hipMemcpy(nextMatrix, cudaNext, sizeof(float) * (rows + 2) * (cols + 2), hipMemcpyDeviceToHost);

    // print matrix

    // for (int i = 1; i < rows + 1; ++i) {
    //     for (int j = 1; j < cols + 1; ++j) {
    //         std::cout << nextMatrix[i * (cols + 2) + j] << "\t\t";
    //     }
    //     std::cout << " " << std::endl;
    // }

    // cal average temperature
    float matrixAvg = calAvg(nextMatrix, rows, cols);

    // temperature difference
    float matrixAvgDiff = calAvgDiff(nextMatrix, matrixAvg, rows, cols);

    return 0;
}